
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#define N 2048
 
// declare the kernel
__global__ void daxpy(double a, double *x, double *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        y[i] += a*x[i];
    }
}
 
int main(void) {
    double *x, *y, *d, a, *dx, *dy;
    double m = -1.;
    double tmp;
    int i;
    size_t size = N*sizeof(double);
 
    x = (double *)malloc(size);
    y = (double *)malloc(size);
 
    // initialize x and y
    srand(time(NULL));
    a = (double)random() / RAND_MAX;
    for (i=0; i<N; i++)
        x[i] = (double)random() / RAND_MAX;
    for (i=0; i<N; i++)
        y[i] = (double)random() / RAND_MAX;
 
    // allocate device memory for x and y
    hipMalloc((void **) &dx, size);
    hipMalloc((void **) &dy, size);
 
    // copy host memory to device memory
    hipMemcpy(dx, x, size, hipMemcpyHostToDevice);
    hipMemcpy(dy, y, size, hipMemcpyHostToDevice);
 
    // launch the kernel function
    daxpy<<<N/64,64>>>(a, dx, dy);
 
    // copy device memory to host memory
    d = (double *)malloc(size);
    hipMemcpy(d, dy, size, hipMemcpyDeviceToHost);
 
    // verify the results
    for (i=0; i<N; i++) {
        y[i] += a * x[i];
	tmp = fabs( (d[i]-y[i])/y[i] );
        if ( tmp > m ) m = tmp;
    }
 
    // deallocate device memory
    hipFree(dx);
    hipFree(dy);
    free(x);
    free(y);
    free(d);
 
    if ( m < 1E-12 ) {
        printf("Success!\n");
	return 0;
    }
    else {
        printf("Failure!\n");
	return 1;
    }
}
